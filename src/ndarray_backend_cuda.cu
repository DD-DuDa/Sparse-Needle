#include "hip/hip_runtime.h"
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iterator>
#include <ostream>
#include <pybind11/numpy.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

#include <hipblas.h>
#include <cusparseLt.h>       // cusparseLt header
#include <thrust/device_vector.h>

#include <iostream>
#include <sstream>

namespace needle {
namespace cuda {

hipblasHandle_t handle;

#define BASE_THREAD_NUM 256

#define TILE 4
typedef float scalar_t;
const size_t ELEM_SIZE = sizeof(scalar_t);
using scalarfn = scalar_t (*)(scalar_t);
using ewisefn = scalar_t (*)(scalar_t, scalar_t);


struct CudaArray {
  CudaArray(const size_t size) {
    hipError_t err = hipMalloc(&ptr, size * ELEM_SIZE);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));
    this->size = size;
  }
  ~CudaArray() { hipFree(ptr); }
  size_t ptr_as_int() { return (size_t)ptr; }
  
  scalar_t* ptr;
  size_t size;
};

struct CudaDims {
  dim3 block, grid;
};

CudaDims CudaOneDim(size_t size) {
  /**
   * Utility function to get cuda dimensions for 1D call
   */
  CudaDims dim;
  size_t num_blocks = (size + BASE_THREAD_NUM - 1) / BASE_THREAD_NUM;
  dim.block = dim3(BASE_THREAD_NUM, 1, 1);
  dim.grid = dim3(num_blocks, 1, 1);
  return dim;
}

#define MAX_VEC_SIZE 8
struct CudaVec {
  uint32_t size;
  uint32_t data[MAX_VEC_SIZE];
};

CudaVec VecToCuda(const std::vector<uint32_t>& x) {
  CudaVec shape;
  if (x.size() > MAX_VEC_SIZE) throw std::runtime_error("Exceeded CUDA supported max dimesions");
  shape.size = x.size();
  for (size_t i = 0; i < x.size(); i++) {
    shape.data[i] = x[i];
  }
  return shape;
}



////////////////////////////////////////////////////////////////////////////////
// Fill call
////////////////////////////////////////////////////////////////////////////////

__global__ void FillKernel(scalar_t* out, scalar_t val, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = val;
}

void Fill(CudaArray* out, scalar_t val) {
  CudaDims dim = CudaOneDim(out->size);
  FillKernel<<<dim.grid, dim.block>>>(out->ptr, val, out->size);
}

////////////////////////////////////////////////////////////////////////////////
// Compact and setitem cals
////////////////////////////////////////////////////////////////////////////////

// Untility function to convert contiguous index i to memory location from strides




__global__ void CompactKernel(const scalar_t* a, scalar_t* out, size_t size, CudaVec shape,
                              CudaVec strides, size_t offset) {
  /**
   * The CUDA kernel for the compact opeation.  This should effectively map a single entry in the 
   * non-compact input a, to the corresponding item (at location gid) in the compact array out.
   * 
   * Args:
   *   a: CUDA pointer to a array
   *   out: CUDA point to out array
   *   size: size of out array
   *   shape: vector of shapes of a and out arrays (of type CudaVec, for past passing to CUDA kernel)
   *   strides: vector of strides of out array
   *   offset: offset of out array
   */
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  /// BEGIN YOUR SOLUTION
  size_t idx = 0, gid_ = gid;
  for (int i = shape.size - 1; i >= 0; i--) {
      idx += (gid_ % shape.data[i]) * strides.data[i];
      gid_ /= shape.data[i];
  }
  if (gid < size) {
      out[gid] = a[offset + idx];
  }
  /// END YOUR SOLUTION
}

void Compact(const CudaArray& a, CudaArray* out, std::vector<uint32_t> shape,
             std::vector<uint32_t> strides, size_t offset) {
  /**
   * Compact an array in memory.  Unlike the C++ version, in CUDA this will primarily call the 
   * relevant CUDA kernel.  In this case, we illustrate how you should set this up (i.e., we give 
   * you the code for this fuction, and also the prototype for the CompactKernel() function).  For
   * the functions after this, however, you'll need to define these kernels as you see fit to 
   * execute the underlying function.
   * 
   * Args:
   *   a: non-compact represntation of the array, given as input
   *   out: compact version of the array to be written
   *   shape: shapes of each dimension for a and out
   *   strides: strides of the *a* array (not out, which has compact strides)
   *   offset: offset of the *a* array (not out, which has zero offset, being compact)
   */

  // Nothing needs to be added here
  CudaDims dim = CudaOneDim(out->size);
  CompactKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size, VecToCuda(shape),
                                         VecToCuda(strides), offset);
}

__global__ void EwiseSetitemKernel(const scalar_t *a, scalar_t *out,
                                   size_t size, CudaVec shape, CudaVec strides,
                                   size_t offset) {
    size_t gid = threadIdx.x + blockDim.x * blockIdx.x;
    size_t idx = 0, gid_ = gid;
    for (int i = shape.size - 1; i >= 0; i--) {
        idx += (gid_ % shape.data[i]) * strides.data[i];
        gid_ /= shape.data[i];
    }
    if (gid < size) {
        out[offset + idx] = a[gid];
    }
}

void EwiseSetitem(const CudaArray& a, CudaArray* out, std::vector<uint32_t> shape,
                  std::vector<uint32_t> strides, size_t offset) {
  /**
   * Set items in a (non-compact) array using CUDA.  Yyou will most likely want to implement a
   * EwiseSetitemKernel() function, similar to those above, that will do the actual work.
   * 
   * Args:
   *   a: _compact_ array whose items will be written to out
   *   out: non-compact array whose items are to be written
   *   shape: shapes of each dimension for a and out
   *   strides: strides of the *out* array (not a, which has compact strides)
   *   offset: offset of the *out* array (not a, which has zero offset, being compact)
   */
  /// BEGIN YOUR SOLUTION
  auto dim = CudaOneDim(a.size);
  EwiseSetitemKernel<<<dim.grid, dim.block>>>(
      a.ptr, out->ptr, a.size, VecToCuda(shape), VecToCuda(strides), offset);
  /// END YOUR SOLUTION
}

__global__ void ScalarSetitemKernel(scalar_t val, scalar_t *out, size_t size,
                                    CudaVec shape, CudaVec strides,
                                    size_t offset) {

    size_t gid = threadIdx.x + blockDim.x * blockIdx.x;
    size_t idx = 0, gid_ = gid;
    for (int i = shape.size - 1; i >= 0; i--) {
        idx += (gid_ % shape.data[i]) * strides.data[i];
        gid_ /= shape.data[i];
    }
    if (gid < size) {
        out[offset + idx] = val;
    }
}

void ScalarSetitem(size_t size, scalar_t val, CudaArray* out, std::vector<uint32_t> shape,
                   std::vector<uint32_t> strides, size_t offset) {
  /**
   * Set items is a (non-compact) array
   * 
   * Args:
   *   size: number of elements to write in out array (note that this will note be the same as
   *         out.size, because out is a non-compact subset array);  it _will_ be the same as the 
   *         product of items in shape, but covenient to just pass it here.
   *   val: scalar value to write to
   *   out: non-compact array whose items are to be written
   *   shape: shapes of each dimension of out
   *   strides: strides of the out array
   *   offset: offset of the out array
   */
  /// BEGIN YOUR SOLUTION
  auto dim = CudaOneDim(size);
  ScalarSetitemKernel<<<dim.grid, dim.block>>>(
      val, out->ptr, size, VecToCuda(shape), VecToCuda(strides), offset);
  /// END YOUR SOLUTION
}

////////////////////////////////////////////////////////////////////////////////
// Elementwise and scalar operations
////////////////////////////////////////////////////////////////////////////////

__global__ void EwiseAddKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] + b[gid];
}

void EwiseAdd(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  /**
   * Add together two CUDA array
   */
  CudaDims dim = CudaOneDim(out->size);
  EwiseAddKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarAddKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] + val;
}

void ScalarAdd(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Add together a CUDA array and a scalar value.
   */
  CudaDims dim = CudaOneDim(out->size);
  ScalarAddKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

/**
 * In the code the follows, use the above template to create analogous elementise
 * and and scalar operators for the following functions.  See the numpy backend for
 * examples of how they should work.
 *   - EwiseMul, ScalarMul
 *   - EwiseDiv, ScalarDiv
 *   - ScalarPower
 *   - EwiseMaximum, ScalarMaximum
 *   - EwiseEq, ScalarEq
 *   - EwiseGe, ScalarGe
 *   - EwiseLog
 *   - EwiseExp
 *   - EwiseTanh
 *
 * If you implement all these naively, there will be a lot of repeated code, so
 * you are welcome (but not required), to use macros or templates to define these
 * functions (however you want to do so, as long as the functions match the proper)
 * signatures above.
 */

////////////////////////////////////////////////////////////////////////////////
// Elementwise and scalar operations
////////////////////////////////////////////////////////////////////////////////

/// BEGIN YOUR SOLUTION
__device__ __forceinline__ scalar_t multiple(scalar_t a, scalar_t b) {
    return a * b;
}
__device__ __forceinline__ scalar_t divide(scalar_t a, scalar_t b) {
    return a / b;
}
__device__ __forceinline__ scalar_t eq(scalar_t a, scalar_t b) {
    return a == b;
}
__device__ __forceinline__ scalar_t ge(scalar_t a, scalar_t b) {
    return a >= b;
}
__device__ __forceinline__ scalar_t max(scalar_t a, scalar_t b) {
    return fmax(a, b);
}
__device__ ewisefn d_mul = multiple;
__device__ ewisefn d_div = divide;
__device__ ewisefn d_eq = eq;
__device__ ewisefn d_ge = ge;
__device__ ewisefn d_max = fmax;
__device__ ewisefn d_pow = powf;
__device__ scalarfn d_exp = expf;
__device__ scalarfn d_tanh = tanhf;
__device__ scalarfn d_log = logf;

__global__ void EwiseOP(const scalar_t *a, scalar_t *out, size_t size,
                        scalarfn op) {
    size_t gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid < size) {

        out[gid] = (*op)(a[gid]);
    }
}

__global__ void EwiseOP(const scalar_t *a, const scalar_t *b, scalar_t *out,
                        size_t size, ewisefn op) {
    size_t gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid < size) {
        out[gid] = (*op)(a[gid], b[gid]);
    }
}

__global__ void ScalarOp(const scalar_t *a, scalar_t val, scalar_t *out,
                         size_t size, ewisefn op) {
    size_t gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid < size) {

        out[gid] = (*op)(a[gid], val);
    }
}

void EwiseMul(const CudaArray &a, const CudaArray &b, CudaArray *out) {
    auto dim = CudaOneDim(out->size);
    ewisefn mul;
    hipMemcpyFromSymbol(&mul, HIP_SYMBOL(d_mul), sizeof(ewisefn));
    EwiseOP<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size, mul);
}

void ScalarMul(const CudaArray &a, scalar_t val, CudaArray *out) {
    auto dim = CudaOneDim(out->size);
    ewisefn mul;
    hipMemcpyFromSymbol(&mul, HIP_SYMBOL(d_mul), sizeof(ewisefn));
    ScalarOp<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size, mul);
}
void EwiseDiv(const CudaArray &a, const CudaArray &b, CudaArray *out) {
    auto dim = CudaOneDim(out->size);
    ewisefn div;
    hipMemcpyFromSymbol(&div, HIP_SYMBOL(d_div), sizeof(ewisefn));
    EwiseOP<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size, div);
}
void ScalarDiv(const CudaArray &a, scalar_t val, CudaArray *out) {
    auto dim = CudaOneDim(out->size);
    ewisefn div;
    hipMemcpyFromSymbol(&div, HIP_SYMBOL(d_div), sizeof(ewisefn));
    ScalarOp<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size, div);
}

void ScalarPower(const CudaArray &a, scalar_t val, CudaArray *out) {
    auto dim = CudaOneDim(out->size);
    ewisefn pow;
    hipMemcpyFromSymbol(&pow, HIP_SYMBOL(d_pow), sizeof(ewisefn));
    ScalarOp<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size, pow);
}

void EwiseMaximum(const CudaArray &a, const CudaArray &b, CudaArray *out) {
    auto dim = CudaOneDim(out->size);
    ewisefn maximum;
    hipMemcpyFromSymbol(&maximum, HIP_SYMBOL(d_max), sizeof(ewisefn));
    EwiseOP<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size,
                                     maximum);
}

void ScalarMaximum(const CudaArray &a, scalar_t val, CudaArray *out) {
    auto dim = CudaOneDim(out->size);
    ewisefn maximum;
    hipMemcpyFromSymbol(&maximum, HIP_SYMBOL(d_max), sizeof(ewisefn));
    ScalarOp<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size, maximum);
}

void EwiseEq(const CudaArray &a, const CudaArray &b, CudaArray *out) {
    auto dim = CudaOneDim(out->size);
    ewisefn eq;
    hipMemcpyFromSymbol(&eq, HIP_SYMBOL(d_eq), sizeof(ewisefn));
    EwiseOP<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size, eq);
}
void ScalarEq(const CudaArray &a, scalar_t val, CudaArray *out) {
    auto dim = CudaOneDim(out->size);
    ewisefn eq;
    hipMemcpyFromSymbol(&eq, HIP_SYMBOL(d_eq), sizeof(ewisefn));
    ScalarOp<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size, eq);
}

void EwiseGe(const CudaArray &a, const CudaArray &b, CudaArray *out) {
    auto dim = CudaOneDim(out->size);
    ewisefn ge;
    hipMemcpyFromSymbol(&ge, HIP_SYMBOL(d_ge), sizeof(ewisefn));
    EwiseOP<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size, ge);
}
void ScalarGe(const CudaArray &a, scalar_t val, CudaArray *out) {
    auto dim = CudaOneDim(out->size);
    ewisefn ge;
    hipMemcpyFromSymbol(&ge, HIP_SYMBOL(d_ge), sizeof(ewisefn));
    ScalarOp<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size, ge);
}

void EwiseLog(const CudaArray &a, CudaArray *out) {
    auto dim = CudaOneDim(out->size);
    scalarfn log;
    hipMemcpyFromSymbol(&log, HIP_SYMBOL(d_log), sizeof(scalarfn));
    EwiseOP<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size, log);
}
void EwiseExp(const CudaArray &a, CudaArray *out) {
    auto dim = CudaOneDim(out->size);
    scalarfn exp;
    hipMemcpyFromSymbol(&exp, HIP_SYMBOL(d_exp), sizeof(scalarfn));
    EwiseOP<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size, exp);
}
void EwiseTanh(const CudaArray &a, CudaArray *out) {
    auto dim = CudaOneDim(out->size);
    scalarfn tanh;
    hipMemcpyFromSymbol(&tanh, HIP_SYMBOL(d_tanh), sizeof(scalarfn));
    EwiseOP<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size, tanh);
}

/// END YOUR SOLUTION
void initCublas() {
    // Create the cuBLAS handle
    hipblasCreate(&handle);
}

void cleanCublas() {
    // Destroy the cuBLAS handle
    hipblasDestroy(handle);
}


__global__ void MatmulKernel_naive(scalar_t *a, scalar_t *b, scalar_t *out,
                                   uint32_t M, uint32_t N, uint32_t P) {
    size_t bidx = blockIdx.x, bidy = blockIdx.y, tidx = threadIdx.x,
           tidy = threadIdx.y;
    auto gidx = bidx * blockDim.x + tidx, gidy = bidy * blockDim.y + tidy;
    if (gidx >= M || gidy >= P) {
        return;
    }
    scalar_t sum = 0.0f;
    for (int i = 0; i < N; i++) {
        sum += a[gidx * N + i] * b[i * P + gidy];
    }
    out[gidx * P + gidy] = sum;
}

__global__ void MatmulKernel_tile(scalar_t *a, scalar_t *b, scalar_t *out,
                                  uint32_t M, uint32_t N, uint32_t P) {
    size_t bidx = blockIdx.x, bidy = blockIdx.y, tidx = threadIdx.x,
           tidy = threadIdx.y;
    int x_range = static_cast<int>(bidx + 1) * TILE - M,
        y_range = static_cast<int>(bidy + 1) * TILE - P;
    if (x_range > 0) {
        a -= x_range * N;
        out -= x_range * P;
    }
    if (y_range > 0) {
        b -= y_range;
        out -= y_range;
    }
    a += bidx * TILE * N;
    b += bidy * TILE;
    out += (bidx * TILE) * P + (bidy * TILE);
    __shared__ scalar_t smemA[TILE][TILE], smemB[TILE][TILE];
    scalar_t accumu = 0.0f;
    for (int i = 0; i < N; i += TILE) {
        smemA[tidx][tidy] = (tidy + i < N) ? a[(tidx)*N + (tidy + i)] : 0.0f;
        smemB[tidx][tidy] = (tidx + i < N) ? b[(tidx + i) * P + tidy] : 0.0f;
        __syncthreads();
        for (int j = 0; j < TILE; j++) {
            accumu += smemA[tidx][j] * smemB[j][tidy];
        }
        __syncthreads();
    }
    out[tidx * P + tidy] = accumu;
}

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
    }                                                                          \
}

int example(void) {
    // Host problem definition, row-major order
    constexpr int m     = 32; // bigger sizes may require dynamic allocations
    constexpr int n     = 32; // bigger sizes may require dynamic allocations
    constexpr int k     = 32; // bigger sizes may require dynamic allocations
    auto          order = HIPSPARSE_ORDER_ROW;
    auto          opA   = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    auto          opB   = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    auto          type  = HIP_R_32F;
    auto          compute_type = CUSPARSE_COMPUTE_TF32;

    bool     is_rowmajor    = (order == HIPSPARSE_ORDER_ROW);
    bool     isA_transposed = (opA != HIPSPARSE_OPERATION_NON_TRANSPOSE);
    bool     isB_transposed = (opB != HIPSPARSE_OPERATION_NON_TRANSPOSE);
    auto     num_A_rows     = (isA_transposed) ? k : m;
    auto     num_A_cols     = (isA_transposed) ? m : k;
    auto     num_B_rows     = (isB_transposed) ? n : k;
    auto     num_B_cols     = (isB_transposed) ? k : n;
    auto     num_C_rows     = m;
    auto     num_C_cols     = n;
    unsigned alignment      = 32;
    auto     lda            = (is_rowmajor) ? num_A_cols : num_A_rows;
    auto     ldb            = (is_rowmajor) ? num_B_cols : num_B_rows;
    auto     ldc            = (is_rowmajor) ? num_C_cols : num_C_rows;
    auto     A_height       = (is_rowmajor) ? num_A_rows : num_A_cols;
    auto     B_height       = (is_rowmajor) ? num_B_rows : num_B_cols;
    auto     C_height       = (is_rowmajor) ? num_C_rows : num_C_cols;
    auto     A_size         = A_height * lda * sizeof(float);
    auto     B_size         = B_height * ldb * sizeof(float);
    auto     C_size         = C_height * ldc * sizeof(float);
    float hA[m * k];
    float hB[k * n];
    float hC[m * n] = {};
    for (int i = 0; i < m * k; i++)
        hA[i] = static_cast<float>(static_cast<float>(std::rand() % 10));
    for (int i = 0; i < k * n; i++)
        hB[i] = static_cast<float>(static_cast<float>(std::rand() % 10));
    float alpha = 1.0f;
    float beta  = 0.0f;
    //--------------------------------------------------------------------------
    // Device memory management
    float *dA, *dB, *dC, *dD, *dA_compressed;
    int    *d_valid;
    CHECK_CUDA( hipMalloc((void**) &dA, A_size) )
    CHECK_CUDA( hipMalloc((void**) &dB, B_size) )
    CHECK_CUDA( hipMalloc((void**) &dC, C_size) )
    CHECK_CUDA( hipMalloc((void**) &d_valid, sizeof(int)) )
    dD = dC;

    CHECK_CUDA( hipMemcpy(dA, hA, A_size, hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB, hB, B_size, hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC, hC, C_size, hipMemcpyHostToDevice) )
    //--------------------------------------------------------------------------
    cusparseLtHandle_t             handle;
    cusparseLtMatDescriptor_t      matA, matB, matC;
    cusparseLtMatmulDescriptor_t   matmul;
    cusparseLtMatmulAlgSelection_t alg_sel;
    cusparseLtMatmulPlan_t         plan;
    hipStream_t                   stream = nullptr;
    CHECK_CUSPARSE( cusparseLtInit(&handle) )
    // matrix descriptor initialization
    CHECK_CUSPARSE( cusparseLtStructuredDescriptorInit(
                                            &handle, &matA, num_A_rows,
                                            num_A_cols, lda, alignment,
                                            type, order,
                                            CUSPARSELT_SPARSITY_50_PERCENT) )
    CHECK_CUSPARSE( cusparseLtDenseDescriptorInit(
                                            &handle, &matB, num_B_rows,
                                            num_B_cols, ldb, alignment,
                                            type, order) )
    CHECK_CUSPARSE( cusparseLtDenseDescriptorInit(
                                            &handle, &matC, num_C_rows,
                                            num_C_cols, ldc, alignment,
                                            type, order) )
    // matmul, algorithm selection, and plan initialization
    CHECK_CUSPARSE( cusparseLtMatmulDescriptorInit(
                                            &handle, &matmul, opA, opB,
                                            &matA, &matB, &matC, &matC,
                                            compute_type) )
    CHECK_CUSPARSE( cusparseLtMatmulAlgSelectionInit(
                                            &handle, &alg_sel, &matmul,
                                            CUSPARSELT_MATMUL_ALG_DEFAULT) )
    int alg = 0;
    CHECK_CUSPARSE( cusparseLtMatmulAlgSetAttribute(
                                            &handle, &alg_sel,
                                            CUSPARSELT_MATMUL_ALG_CONFIG_ID,
                                            &alg, sizeof(alg)))
    size_t workspace_size;
    CHECK_CUSPARSE( cusparseLtMatmulPlanInit(&handle, &plan, &matmul, &alg_sel,
                                             workspace_size) )
    //--------------------------------------------------------------------------
    // Prune the A matrix (in-place) and check the correctness

    CHECK_CUSPARSE( cusparseLtSpMMAPrune(&handle, &matmul, dA, dA,
                                         CUSPARSELT_PRUNE_SPMMA_TILE, stream) )
    int is_valid;
    CHECK_CUSPARSE( cusparseLtSpMMAPruneCheck(&handle, &matmul, dA,
                                              &is_valid, stream) )

    CHECK_CUDA( hipMemcpyAsync(&is_valid, d_valid, sizeof(int),
                                hipMemcpyDeviceToHost, stream) )
    CHECK_CUDA( hipStreamSynchronize(stream) )
    std::cout << is_valid << std::endl;
    if (is_valid != 0) {
        std::printf("!!!! The matrix has been pruned in a wrong way. "
                    "cusparseLtMatmul will not provide correct results\n");
        return EXIT_FAILURE;
    }
    //--------------------------------------------------------------------------
    // Compress the A matrix
    size_t compressed_size;
    CHECK_CUSPARSE( cusparseLtSpMMACompressedSize(&handle, &plan,
                                                  &compressed_size) )
    CHECK_CUDA( hipMalloc((void**) &dA_compressed, compressed_size) )

    CHECK_CUSPARSE( cusparseLtSpMMACompress(&handle, &plan, dA,
                                            dA_compressed, stream) )
    //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    // Search the best kernel
    void*         d_workspace = nullptr;
    int           num_streams = 0;
    hipStream_t* streams     = nullptr;
    CHECK_CUSPARSE( cusparseLtMatmulSearch(&handle, &plan, &alpha,
                                           dA_compressed, dB, &beta,
                                           dC, dD, d_workspace,
                                           streams, num_streams) )
    int alg_id;
    CHECK_CUSPARSE( cusparseLtMatmulAlgGetAttribute(
                                           &handle, &alg_sel,
                                           CUSPARSELT_MATMUL_ALG_CONFIG_ID,
                                           &alg_id, sizeof(alg_id)) )
    int32_t splitK, splitKBuffers;
    cusparseLtSplitKMode_t splitKMode;

    CHECK_CUSPARSE( cusparseLtMatmulAlgGetAttribute(
                                           &handle, &alg_sel,
                                           CUSPARSELT_MATMUL_SPLIT_K,
                                           &splitK, sizeof(splitK)) )

    CHECK_CUSPARSE( cusparseLtMatmulAlgGetAttribute(
                                           &handle, &alg_sel,
                                           CUSPARSELT_MATMUL_SPLIT_K_MODE,
                                           &splitKMode, sizeof(splitKMode)) )

    CHECK_CUSPARSE( cusparseLtMatmulAlgGetAttribute(
                                           &handle, &alg_sel,
                                           CUSPARSELT_MATMUL_SPLIT_K_BUFFERS,
                                           &splitKBuffers,
                                           sizeof(splitKBuffers)) )
    //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    CHECK_CUSPARSE( cusparseLtMatmulPlanInit(&handle, &plan, &matmul, &alg_sel,
                                             workspace_size) )

    CHECK_CUSPARSE( cusparseLtMatmulGetWorkspace(&handle, &plan,
                                                 &workspace_size))

    CHECK_CUDA( hipMalloc((void**)&d_workspace, workspace_size) )
    // Perform the matrix multiplication
    CHECK_CUSPARSE( cusparseLtMatmul(&handle, &plan, &alpha, dA_compressed, dB,
                                     &beta, dC, dD, d_workspace, streams,
                                     num_streams) )
    //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    // destroy plan and handle
    CHECK_CUSPARSE( cusparseLtMatDescriptorDestroy(&matA) )
    CHECK_CUSPARSE( cusparseLtMatDescriptorDestroy(&matB) )
    CHECK_CUSPARSE( cusparseLtMatDescriptorDestroy(&matC) )
    CHECK_CUSPARSE( cusparseLtMatmulPlanDestroy(&plan) )
    CHECK_CUSPARSE( cusparseLtDestroy(&handle) )
    //--------------------------------------------------------------------------
    // device result check
    // matrix A has been pruned
    CHECK_CUDA( hipMemcpy(hA, dA, A_size, hipMemcpyDeviceToHost) )
    CHECK_CUDA( hipMemcpy(hC, dC, C_size, hipMemcpyDeviceToHost) )

    bool A_std_layout = (is_rowmajor != isA_transposed);
    bool B_std_layout = (is_rowmajor != isB_transposed);
    // host computation
    float hC_result[m * n];
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            float sum  = 0.0f;
            for (int k1 = 0; k1 < k; k1++) {
                auto posA = (A_std_layout) ? i * lda + k1 : i + k1 * lda;
                auto posB = (B_std_layout) ? k1 * ldb + j : k1 + j * ldb;
                sum      += static_cast<float>(hA[posA]) *  // [i][k]
                            static_cast<float>(hB[posB]);   // [k][j]
            }
            auto posC       = (is_rowmajor) ? i * ldc + j : i + j * ldc;
            hC_result[posC] = sum;  // [i][j]
        }
    }
    // host-device comparison
    int correct = 1;
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            auto pos          = (is_rowmajor) ? i * ldc + j : i + j * ldc;
            auto device_value = static_cast<float>(hC[pos]);
            auto host_value   = hC_result[pos];
            if (device_value != host_value) {
                // direct floating point comparison is not reliable
                std::printf("(%d, %d):\t%f vs. %f\n",
                            i, j, host_value, device_value);
                correct = 0;
                break;
            }
        }
    }
    if (correct)
        std::printf("spmma_example test PASSED\n");
    else
        std::printf("spmma_example test FAILED: wrong result\n");
    //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK_CUDA( hipFree(dA_compressed) )
    CHECK_CUDA( hipFree(dA) )
    CHECK_CUDA( hipFree(dB) )
    CHECK_CUDA( hipFree(dC) )
    CHECK_CUDA( hipFree(d_valid) )
    CHECK_CUDA( hipFree(d_workspace) )
    return EXIT_SUCCESS;
}

template <typename type_t>
void cusparsespmma(
    type_t* dA,
    type_t* dB,
    type_t* dC,
    size_t m,
    size_t n,
    size_t k,
    hipsparseOperation_t opA = HIPSPARSE_OPERATION_NON_TRANSPOSE,
    hipsparseOperation_t opB = HIPSPARSE_OPERATION_NON_TRANSPOSE,
    float alpha = 1.0f,
    float beta = 0.0f) {

    // Host problem definition, row-major order
    auto          order = HIPSPARSE_ORDER_COL;
    auto          type  = HIP_R_32F;
    auto          compute_type = CUSPARSE_COMPUTE_TF32;

    bool     is_rowmajor    = (order == HIPSPARSE_ORDER_ROW);
    bool     isA_transposed = (opA != HIPSPARSE_OPERATION_NON_TRANSPOSE);
    bool     isB_transposed = (opB != HIPSPARSE_OPERATION_NON_TRANSPOSE);
    auto     num_A_rows     = (isA_transposed) ? k : m;
    auto     num_A_cols     = (isA_transposed) ? m : k;
    auto     num_B_rows     = (isB_transposed) ? n : k;
    auto     num_B_cols     = (isB_transposed) ? k : n;
    auto     num_C_rows     = m;
    auto     num_C_cols     = n;
    unsigned alignment      = 32;
    auto     lda            = (is_rowmajor) ? num_A_cols : num_A_rows;
    auto     ldb            = (is_rowmajor) ? num_B_cols : num_B_rows;
    auto     ldc            = (is_rowmajor) ? num_C_cols : num_C_rows;


    //--------------------------------------------------------------------------
    // Device memory management
    float *dD, *dA_compressed;
    dD = dC;

    //--------------------------------------------------------------------------
    cusparseLtHandle_t             handle;
    cusparseLtMatDescriptor_t      matA, matB, matC;
    cusparseLtMatmulDescriptor_t   matmul;
    cusparseLtMatmulAlgSelection_t alg_sel;
    cusparseLtMatmulPlan_t         plan;
    hipStream_t                   stream = nullptr;
    CHECK_CUSPARSE( cusparseLtInit(&handle) )
    // matrix descriptor initialization
    CHECK_CUSPARSE( cusparseLtStructuredDescriptorInit(
                                            &handle, &matA, num_A_rows,
                                            num_A_cols, lda, alignment,
                                            type, order,
                                            CUSPARSELT_SPARSITY_50_PERCENT) )
    CHECK_CUSPARSE( cusparseLtDenseDescriptorInit(
                                            &handle, &matB, num_B_rows,
                                            num_B_cols, ldb, alignment,
                                            type, order) )
    CHECK_CUSPARSE( cusparseLtDenseDescriptorInit(
                                            &handle, &matC, num_C_rows,
                                            num_C_cols, ldc, alignment,
                                            type, order) )
    // matmul, algorithm selection, and plan initialization
    CHECK_CUSPARSE( cusparseLtMatmulDescriptorInit(
                                            &handle, &matmul, opA, opB,
                                            &matA, &matB, &matC, &matC,
                                            compute_type) )
    CHECK_CUSPARSE( cusparseLtMatmulAlgSelectionInit(
                                            &handle, &alg_sel, &matmul,
                                            CUSPARSELT_MATMUL_ALG_DEFAULT) )

    size_t workspace_size;
    CHECK_CUSPARSE( cusparseLtMatmulPlanInit(&handle, &plan, &matmul, &alg_sel,
                                             workspace_size) )
    //--------------------------------------------------------------------------
    // Prune the A matrix (in-place) and check the correctness
    CHECK_CUSPARSE( cusparseLtSpMMAPrune(&handle, &matmul, dA, dA,
                                         CUSPARSELT_PRUNE_SPMMA_TILE, stream) )

    //--------------------------------------------------------------------------
    // Compress the A matrix
    size_t compressed_size;
    CHECK_CUSPARSE( cusparseLtSpMMACompressedSize(&handle, &plan,
                                                  &compressed_size) )
    CHECK_CUDA( hipMalloc((void**) &dA_compressed, compressed_size) )

    CHECK_CUSPARSE( cusparseLtSpMMACompress(&handle, &plan, dA,
                                            dA_compressed, stream) )
    //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    // Search the best kernel
    void*         d_workspace = nullptr;
    int           num_streams = 0;
    hipStream_t* streams     = nullptr;


    CHECK_CUSPARSE( cusparseLtMatmulGetWorkspace(&handle, &plan,
                                                 &workspace_size))

    CHECK_CUDA( hipMalloc((void**)&d_workspace, workspace_size) )
    // Perform the matrix multiplication
    CHECK_CUSPARSE( cusparseLtMatmul(&handle, &plan, &alpha, dA_compressed, dB,
                                     &beta, dC, dD, d_workspace, streams,
                                     num_streams) )
    //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    // destroy plan and handle
    CHECK_CUSPARSE( cusparseLtMatDescriptorDestroy(&matA) )
    CHECK_CUSPARSE( cusparseLtMatDescriptorDestroy(&matB) )
    CHECK_CUSPARSE( cusparseLtMatDescriptorDestroy(&matC) )
    CHECK_CUSPARSE( cusparseLtMatmulPlanDestroy(&plan) )
    CHECK_CUSPARSE( cusparseLtDestroy(&handle) )

    //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK_CUDA( hipFree(dA_compressed) )
    CHECK_CUDA( hipFree(d_workspace) )

}

void Matmul(const CudaArray& a, const CudaArray& b, CudaArray* out, uint32_t M, uint32_t N,
            uint32_t P) {
  /**
   * Multiply two (compact) matrices into an output (also comapct) matrix.  You will want to look
   * at the lecture and notes on GPU-based linear algebra to see how to do this.  Since ultimately
   * mugrade is just evaluating correctness, you _can_ implement a version that simply parallelizes
   * over (i,j) entries in the output array.  However, to really get the full benefit of this
   * problem, we would encourage you to use cooperative fetching, shared memory register tiling, 
   * and other ideas covered in the class notes.  Note that unlike the tiled matmul function in
   * the CPU backend, here you should implement a single function that works across all size
   * matrices, whether or not they are a multiple of a tile size.  As with previous CUDA
   * implementations, this function here will largely just set up the kernel call, and you should
   * implement the logic in a separate MatmulKernel() call.
   * 
   *
   * Args:
   *   a: compact 2D array of size m x n
   *   b: comapct 2D array of size n x p
   *   out: compact 2D array of size m x p to write the output to
   *   M: rows of a / out
   *   N: columns of a / rows of b
   *   P: columns of b / out
   */
    /********************* SPARSE ***************************/
    /********************* SPARSE ***************************/

  /// BEGIN YOUR SOLUTION
    // std::cout << std::endl << "Matmul in cuda" << std::endl;

    // Fill(out, 0.0f);
    // if (M < TILE || P < TILE || N < TILE) {
    //     dim3 block(TILE, TILE);
    //     dim3 grid((M - 1) / TILE + 1, (P - 1) / TILE + 1);
    //     MatmulKernel_naive<<<grid, block>>>(a.ptr, b.ptr, out->ptr, M, N, P);
    // } else {
    //     dim3 block(TILE, TILE);
    //     dim3 grid((M - 1) / TILE + 1, (P - 1) / TILE + 1);
    //     MatmulKernel_tile<<<grid, block>>>(a.ptr, b.ptr, out->ptr, M, N, P);
    // }
    // hipDeviceSynchronize();
  /// END YOUR SOLUTION

    std::cout << M << ' ' << N << ' ' << P << std::endl;
    if (N % 8 == 0 && P % 8 == 0) {
        // sparse tc
        cusparsespmma<float>(b.ptr, a.ptr, out->ptr, P, M, N);
    }
    else {
        // cublas
        
        // hipblasHandle_t cublas_handle;
        // hipblasCreate(&cublas_handle);
        float cublas_alpha = 1.0f;
        float cublas_beta = 0.0f;

        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, P, M, N, &cublas_alpha, b.ptr, P, a.ptr, N, &cublas_beta, out->ptr, P);
        // hipblasDestroy(cublas_handle);
    }


}


////////////////////////////////////////////////////////////////////////////////
// Max and sum reductions
////////////////////////////////////////////////////////////////////////////////
__global__ void ReduceMaxKernel(const scalar_t *a, scalar_t *out,
                                size_t reduce_size, size_t len) {
    size_t gid = threadIdx.x + blockDim.x * blockIdx.x;
    if (gid >= len) {
        return;
    }
    scalar_t maxValue = a[gid * reduce_size];
    for (size_t i = gid * reduce_size; i < (gid + 1) * reduce_size; i++) {
        maxValue = max(maxValue, a[i]);
    }

    out[gid] = maxValue;
}

void ReduceMax(const CudaArray& a, CudaArray* out, size_t reduce_size) {
  /**
   * Reduce by taking maximum over `reduce_size` contiguous blocks.  Even though it is inefficient,
   * for simplicity you can perform each reduction in a single CUDA thread.
   * 
   * Args:
   *   a: compact array of size a.size = out.size * reduce_size to reduce over
   *   out: compact array to write into
   *   redice_size: size of the dimension to reduce over
   */
  /// BEGIN YOUR SOLUTION
  auto dim = CudaOneDim(out->size);
  ReduceMaxKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, reduce_size,
                                           out->size);
  /// END YOUR SOLUTION
}


__global__ void ReduceSumKernel(const scalar_t *a, scalar_t *out,
                                size_t reduce_size, size_t len) {
    size_t gid = threadIdx.x + blockDim.x * blockIdx.x;
    scalar_t sum = 0.0f;
    if (gid >= len) {
        return;
    }
    for (size_t i = gid * reduce_size; i < (gid + 1) * reduce_size; i++) {
        sum += a[i];
    }
    out[gid] = sum;
}

void ReduceSum(const CudaArray& a, CudaArray* out, size_t reduce_size) {
  /**
   * Reduce by taking summation over `reduce_size` contiguous blocks.  Again, for simplicity you 
   * can perform each reduction in a single CUDA thread.
   * 
   * Args:
   *   a: compact array of size a.size = out.size * reduce_size to reduce over
   *   out: compact array to write into
   *   redice_size: size of the dimension to reduce over
   */
  /// BEGIN YOUR SOLUTION
  auto dim = CudaOneDim(out->size);
  ReduceSumKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, reduce_size,
                                           out->size);
  /// END YOUR SOLUTION
}

}  // namespace cuda
}  // namespace needle

PYBIND11_MODULE(ndarray_backend_cuda, m) {
  namespace py = pybind11;
  using namespace needle;
  using namespace cuda;

  m.attr("__device_name__") = "cuda";
  m.attr("__tile_size__") = TILE;

  py::class_<CudaArray>(m, "Array")
      .def(py::init<size_t>(), py::return_value_policy::take_ownership)
      .def_readonly("size", &CudaArray::size)
      .def("ptr", &CudaArray::ptr_as_int);

  // return numpy array, copying from CPU
  m.def("to_numpy", [](const CudaArray& a, std::vector<size_t> shape, std::vector<size_t> strides,
                       size_t offset) {
    std::vector<size_t> numpy_strides = strides;
    std::transform(numpy_strides.begin(), numpy_strides.end(), numpy_strides.begin(),
                   [](size_t& c) { return c * ELEM_SIZE; });

    // copy memory to host
    scalar_t* host_ptr = (scalar_t*)std::malloc(a.size * ELEM_SIZE);
    if (host_ptr == 0) throw std::bad_alloc();
    hipError_t err = hipMemcpy(host_ptr, a.ptr, a.size * ELEM_SIZE, hipMemcpyDeviceToHost);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));

    // return numpy array
    py::capsule deallocate_buffer(host_ptr, [](void* p) { free(p); });
    return py::array_t<scalar_t>(shape, numpy_strides, host_ptr + offset, deallocate_buffer);
  });

  // copy numpy array to GPU
  m.def("from_numpy", [](py::array_t<scalar_t> a, CudaArray* out) {
    hipError_t err =
        hipMemcpy(out->ptr, a.request().ptr, out->size * ELEM_SIZE, hipMemcpyHostToDevice);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));
  });

  m.def("fill", Fill);
  m.def("compact", Compact);
  m.def("ewise_setitem", EwiseSetitem);
  m.def("scalar_setitem", ScalarSetitem);
  m.def("ewise_add", EwiseAdd);
  m.def("scalar_add", ScalarAdd);

  m.def("ewise_mul", EwiseMul);
  m.def("scalar_mul", ScalarMul);
  m.def("ewise_div", EwiseDiv);
  m.def("scalar_div", ScalarDiv);
  m.def("scalar_power", ScalarPower);

  m.def("ewise_maximum", EwiseMaximum);
  m.def("scalar_maximum", ScalarMaximum);
  m.def("ewise_eq", EwiseEq);
  m.def("scalar_eq", ScalarEq);
  m.def("ewise_ge", EwiseGe);
  m.def("scalar_ge", ScalarGe);

  m.def("ewise_log", EwiseLog);
  m.def("ewise_exp", EwiseExp);
  m.def("ewise_tanh", EwiseTanh);

  m.def("init_cublas", initCublas);
  m.def("clean_cublas", cleanCublas);
  m.def("matmul", Matmul);

  m.def("reduce_max", ReduceMax);
  m.def("reduce_sum", ReduceSum);
}
